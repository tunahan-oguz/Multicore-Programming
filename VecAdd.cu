#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
// MAX THREADS ALLOWED PER BLOCK IS 1024 IN MY SYSTEM
#define THRDS 1024

__global__ void vecAdd(float* A, float* B, float* C, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N)
        C[i] = A[i] + B[i];
}

int main(int argc, char const *argv[]){
    int N = atoi(argv[1]);
    float* A = new float[N];
    float* B = new float[N];
    float* C = new float[N];
    for (size_t i = 0; i < N; i++){
        A[i] = 1;
        B[i] = 2;
        C[i] = 0;
    }
    float* cu_A;
    float* cu_B;
    float* cu_C;
    hipMalloc((void**)&cu_A, N * sizeof(float));
    hipMalloc((void**)&cu_B, N * sizeof(float));
    hipMalloc((void**)&cu_C, N * sizeof(float));
    hipMemcpy(cu_A, A, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(cu_B, B, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemset(cu_C, 0, sizeof(float) * N);

    int numberOfBlocks = (N + THRDS - 1) / THRDS;
    int threadsPerBlock = THRDS;

    vecAdd<<<numberOfBlocks, threadsPerBlock>>>(cu_A, cu_B, cu_C, N);

    hipMemcpy(C, cu_C, sizeof(float) * N, hipMemcpyDeviceToHost);
    hipFree(cu_C);
    hipFree(cu_A);
    hipFree(cu_B);
    
    // for (size_t i = 0; i < N; i++){
    //     cout << C[i]<< " ";
    // }
    // cout << endl;

    return 0;
}
