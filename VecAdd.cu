#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void vecAdd(float* A, float* B, float* C, int N){
    int i = threadIdx.x;
    // if(i < N)
        C[i] = A[i] + B[i];
}

int main(int argc, char const *argv[]){
    int N = 4096;
    float* A = new float[N];
    float* B = new float[N];
    float* C = new float[N];
    for (size_t i = 0; i < N; i++){
        A[i] = 1;
        B[i] = 2;
        C[i] = 0;
    }
    float* cu_A;
    float* cu_B;
    float* cu_C;
    hipMalloc((void**)&cu_A, N * sizeof(float));
    hipMalloc((void**)&cu_B, N * sizeof(float));
    hipMalloc((void**)&cu_C, N * sizeof(float));
    hipMemcpy(cu_A, A, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(cu_B, B, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemset(cu_C, 0, sizeof(float) * N);

    int numberOfBlocks = 1;
    int numberOfThreads = N;

    vecAdd<<<numberOfBlocks, numberOfThreads>>>(cu_A, cu_B, cu_C, N);

    hipMemcpy(C, cu_C, sizeof(float) * N, hipMemcpyDeviceToHost);
    hipFree(cu_C);
    hipFree(cu_A);
    hipFree(cu_B);
    

    return 0;
}
