#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "mat_mul/MatMul.hpp"
#include "Report.hpp"

using namespace std;

#define TILE_WIDTH 128


__global__ void matMul(float* A, float* B, float* C, int w){
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    float sum = 0;
    for (size_t i = 0; i < w; i++){
        sum += A[row * w + i] * B[i * w + col];
    }
    C[row * w + col] = sum;
}


int main(int argc, char const *argv[]){
    int matrix_W = 4096 * 4;
    int size = matrix_W * matrix_W; 
    float* Mat1 = new float[size]; 
    float* Mat2 = new float[size]; 
    float* res = new float[size]; 
    for (size_t i = 0; i < matrix_W; i++){
        for (size_t j = 0; j < matrix_W; j++){
            Mat1[i* matrix_W + j] = 1;
            Mat2[i* matrix_W + j] = 1;
            
        }
    }
    
    // for (size_t i = 0; i < matrix_W; i++){
    //     for (size_t j = 0; j < matrix_W; j++){
    //         cout << Mat1[i* matrix_W + j] << " ";            
    //     }
    //     cout << endl;
    // }
    

    float* A;
    float* B;
    float* C;

    auto alloc_start = chrono::high_resolution_clock::now();
    hipMalloc((void**)&A, sizeof(float) * size);
    hipMalloc((void**)&B, sizeof(float) * size);
    hipMalloc((void**)&C, sizeof(float) * size);
    auto alloc_finish = chrono::high_resolution_clock::now();
    hipMemcpy(A, Mat1, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(B, Mat2, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemset(C, 0, sizeof(float) * size);
    auto cpy_finish = chrono::high_resolution_clock::now();

    dim3 gridDim(matrix_W / TILE_WIDTH, matrix_W / TILE_WIDTH);
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
    
    auto mul_start = chrono::high_resolution_clock::now();
    matMul<<<gridDim, blockDim>>>(A, B, C, matrix_W);
    auto mul_finish = chrono::high_resolution_clock::now();    
    hipMemcpy(res, C, sizeof(float) * size, hipMemcpyDeviceToHost);
    auto copy_host_finish = chrono::high_resolution_clock::now();

    hipFree(C);
    hipFree(A);
    hipFree(B);
    auto free_finish = chrono::high_resolution_clock::now();

    auto alloc_duration = chrono::duration_cast<chrono::microseconds>(alloc_finish - alloc_start);
    auto copy_host2device_duration = chrono::duration_cast<chrono::microseconds>(cpy_finish - alloc_finish);
    auto procces_dur = chrono::duration_cast<chrono::microseconds>(mul_finish - mul_start);
    auto copy_back = chrono::duration_cast<chrono::microseconds>(copy_host_finish - mul_finish);
    auto total = chrono::duration_cast<chrono::microseconds>(free_finish - alloc_start);

    cout<< "Total process "<< total.count() << endl;
    cout<< "Allocation "<< alloc_duration.count() << endl;
    cout<< "Copy to device "<< copy_host2device_duration.count() << endl;
    cout<< "Multiplication "<< procces_dur.count() << endl;
    cout<< "copy_back "<< copy_back.count() << endl;




    // for (size_t i = 0; i < matrix_W; i++){
    //     for (size_t j = 0; j < matrix_W; j++){
    //         cout << res[i* matrix_W + j] << " ";            
    //     }
    //     cout << endl;
    // }

    return 0;
}
