#include "hip/hip_runtime.h"
#include "Hist.h"

using namespace std;

__global__ void kernel(uint8_t* image, int* d_hist, int im_size);


CudaCalculator::CudaCalculator(uint8_t* image, int y, int x){
    this->image = image;
    this->x = x;
    this->y = y;
    this->histogram = new int[256];
}

int* CudaCalculator::operator()(){
    // allocate device memory
    uint8_t* cuda_image;
    hipMalloc((void**)&cuda_image, x * y * sizeof(uint8_t));
    hipMalloc((void**)&d_histogram, 256 * sizeof(int));
    hipMemcpy(cuda_image, image, x * y * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemset(d_histogram, 0, 256 * sizeof(int));
    //launch kernel
    int threadsPerBlock = 256;
    int numberOfBlocks = (x * y + threadsPerBlock - 1) / threadsPerBlock;
    kernel<<<numberOfBlocks, threadsPerBlock>>>(cuda_image, d_histogram, x * y);
    // copy results to host
    hipMemcpy(histogram, d_histogram, 256 * sizeof(int), hipMemcpyDeviceToHost);
    // free device mem
    hipFree(cuda_image);
    hipFree(d_histogram);
    return this->histogram;
}

__global__ void kernel(uint8_t* image, int* d_hist, int im_size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < im_size) {
        atomicAdd(&d_hist[image[tid]], 1);
    }
}