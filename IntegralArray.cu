#include <hip/hip_runtime.h>
#include <iostream>


using namespace std;
// MAX numberOfBlocks allowed in my system is 128
#define NBLOCKS 128

__global__ void integral_array(int* src, int* res, int N){
    int numel_per_block = N / blockDim.x;
    int start = blockIdx.x * numel_per_block + threadIdx.x;
    int end = start + numel_per_block;
    if (start < N){
        res[start] = src[start];
        for (size_t i = start + 1; i < end; i++){
            res[i] = res[i - 1] + src[i];
        }
    }
    
}

int main(int argc, char const *argv[]){

    int N = atoi(argv[1]);
    int* A = new int[N];
    int* res = new int[N];

    for (size_t i = 0; i < N; i++){
        A[i] = 1;
        res[i] = 0;
    }
    int* A_device;
    int* res_device;

    hipMalloc((void**)&A_device, sizeof(int) * N);
    hipMalloc((void**)&res_device, sizeof(int) * N);
    hipMemset(res_device, 0, sizeof(int) * N);
    hipMemcpy(A_device, A, sizeof(int) * N, hipMemcpyHostToDevice);
    
    int numberOfBlocks = NBLOCKS;
    int threadsPerBlock = (N + NBLOCKS - 1) / NBLOCKS;

    integral_array<<<numberOfBlocks, threadsPerBlock>>>(A_device, res_device, N);

    hipMemcpy(res, res_device, sizeof(int) * N, hipMemcpyDeviceToHost);

    hipFree(res_device);
    hipFree(A);

    for (size_t i = 0; i < N; i++){
        cout << res[i]<< " ";
    }
    cout << endl;

    return 0;
}
