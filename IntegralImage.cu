#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>


using namespace std;

__global__ void kernel(uint8_t* image, long* dest, int N){
    int start_idx = blockIdx.x * N;
    dest[start_idx] = image[start_idx];
    for (size_t i = 1; i < N; i++){
        dest[start_idx + i] = dest[start_idx + i - 1] + image[start_idx + i];
    }
    __syncthreads();
    start_idx = blockIdx.x;
    for (size_t i = 1; i < N; i++){
        dest[i * N + start_idx] = dest[(i - 1) * N + start_idx] + dest[i * N + start_idx];
    }
}

int main(int argc, char const *argv[]){
    int N = 25600;
    uint8_t* image = new uint8_t[N * N];
    long* output = new long[N * N];
    for (size_t i = 0; i < N * N; i++){
        image[i] = 1;
        output[i] = 0;
    }
    auto start = chrono::high_resolution_clock::now();
    uint8_t* image_dev;
    long* out_dev;
    hipMalloc((void**)&image_dev, sizeof(uint8_t) * N * N);
    hipMemcpy(image_dev, image, sizeof(uint8_t) * N * N, hipMemcpyHostToDevice);
    hipMalloc((void**)&out_dev, sizeof(long) * N * N);
    hipMemcpy(out_dev, output, sizeof(long) * N * N, hipMemcpyHostToDevice);

    int numberOfBlocks = N;
    int threadsPerBlock = 1;
    auto kernel_start = chrono::high_resolution_clock::now();
    kernel<<<numberOfBlocks, threadsPerBlock>>>(image_dev, out_dev, N);
    auto kernel_end = chrono::high_resolution_clock::now();
    auto kernel_duration = chrono::duration_cast<chrono::microseconds>(kernel_end - kernel_start);
    cout << "Calculated in " << kernel_duration.count() << " microseconds." << endl;

    hipMemcpy(output, out_dev, sizeof(long) * N * N, hipMemcpyDeviceToHost);
    hipFree(out_dev);
    hipFree(image_dev);
    auto end = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::microseconds>(end - start);
    cout << "Integral image in " << duration.count() << " microseconds." << endl;
    // for (size_t i = 0; i < N; i++){
    //     for (size_t j = 0; j < N; j++){
    //         cout << output[i* N + j] << " ";
    //     }
    //     cout << endl;
    // }
    // cout << endl;

    return 0;
}
